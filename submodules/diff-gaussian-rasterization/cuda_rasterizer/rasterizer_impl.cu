#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <vector>
#include <tuple>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"
#include "gsgn.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	int* radii,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

        getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{			
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
                gaussian_keys_unsorted[off] = key;
                gaussian_values_unsorted[off] = idx;
                off++;
            }
        }
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;

	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

CudaRasterizer::BinningStateReduced CudaRasterizer::BinningStateReduced::fromChunk(char*& chunk, size_t P)
{
	BinningStateReduced binning;
	obtain(chunk, binning.point_list, P, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
std::tuple<int, int64_t> CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
    const float cx, float cy,
	const bool prefiltered,
	float* out_color,
    int* n_contrib_vol_rend,
	bool* is_gaussian_hit,
    int* radii,
	bool debug)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + GSGN_BLOCK_X - 1) / GSGN_BLOCK_X, (height + GSGN_BLOCK_Y - 1) / GSGN_BLOCK_Y, 1);
	dim3 block(GSGN_BLOCK_X, GSGN_BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (GSGN_NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
        cx, cy,
		radii,
		geomState.means2D,
		geomState.depths,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		tile_grid,
		geomState.tiles_touched,
		prefiltered
	), debug)

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

    // Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

    // Create BinningState and also remember how many bytes are needed for point_list, because that is the only thing needed afterwards (for backward or GN)
    // we can then resize the tensor using this information and reduce the memory by 90%
	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
    char* begin_binning_point_list = binning_chunkptr;
    char* end_binning_point_list = binning_chunkptr;
    BinningStateReduced::fromChunk(end_binning_point_list, num_rendered);
    int64_t num_bytes_binning_point_list = end_binning_point_list - begin_binning_point_list;
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid)
	CHECK_CUDA(, debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
        binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)

	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
            binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
        binningState.point_list,
		width, height,
		geomState.means2D,
		feature_ptr,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
        n_contrib_vol_rend,
        is_gaussian_hit,
		background,
		out_color), debug)

	return std::make_tuple(num_rendered, num_bytes_binning_point_list);
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
    const float cx, float cy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningStateReduced binningState = BinningStateReduced::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + GSGN_BLOCK_X - 1) / GSGN_BLOCK_X, (height + GSGN_BLOCK_Y - 1) / GSGN_BLOCK_Y, 1);
	const dim3 block(GSGN_BLOCK_X, GSGN_BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor), debug)

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
        cx, cy,
        width, height,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot), debug)
}

// -----------------------------------
// ADDITIONAL METHODS FOR GN SUPPORT |
// -----------------------------------

void CudaRasterizer::PackedGSGNDataSpec::allocate_pointer_memory() {
    assert(! pointers_changed);

    hipMallocManaged((void**) &point_list_ptrs, num_images * sizeof(uint32_t*));
    hipMallocManaged((void**) &ranges_ptrs, num_images * sizeof(uint2*));
    hipMallocManaged((void**) &n_contrib_ptrs, num_images * sizeof(int32_t*));
    hipMallocManaged((void**) &accum_alpha_ptrs, num_images * sizeof(float*));

    for(int i=0; i < num_images; i++) {
        BinningStateReduced binningState = BinningStateReduced::fromChunk(binningBuffer_ptrs[i], num_rendered[i]);
        point_list_ptrs[i] = binningState.point_list;

        ImageState imgState = ImageState::fromChunk(imageBuffer_ptrs[i], W * H);
        ranges_ptrs[i] = imgState.ranges;
        n_contrib_ptrs[i] = imgState.n_contrib;
        accum_alpha_ptrs[i] = imgState.accum_alpha;
    }

    pointers_changed = true;
}

void CudaRasterizer::Rasterizer::reorder_geometry_buffer(const int P, char* geom_buffer, int* radii, std::function<char* (size_t)> out_geom_buffer) {
    // cast char* to GeometryState
    GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
    if (radii == nullptr) {
		radii = geomState.internal_radii;
	}

    // resize required bytes for GeometryStateReduced
    // chunkptr is a GPU pointer to the start of the memory
    size_t chunk_size = P * sizeof(GeometryStateReduced);
	char* chunkptr = out_geom_buffer(chunk_size);

    // fill it
    CHECK_CUDA(CudaRasterizer::GSGN::fill_reordered_geometry_buffer(
        P,
        geomState.clamped,
        radii,
        geomState.means2D,
        geomState.cov3D,
        geomState.conic_opacity,
        geomState.rgb,
        chunkptr
    ), false)

}

void CudaRasterizer::Rasterizer::filter_reordered_geometry_buffer(const int num_visible_gaussians, int* map_cache_to_gaussians, char* geom_buffer, std::function<char* (size_t)> out_geom_buffer) {
    // resize required bytes for GeometryStateReduced
    // chunkptr is a GPU pointer to the start of the memory
    size_t chunk_size = num_visible_gaussians * sizeof(GeometryStateReduced);
	char* chunkptr = out_geom_buffer(chunk_size);

    // fill it
    CHECK_CUDA(CudaRasterizer::GSGN::filter_reordered_geometry_buffer(
        num_visible_gaussians,
        map_cache_to_gaussians,
        geom_buffer,
        chunkptr
    ), false)
}

// eval_jtf_and_get_sparse_jacobian
template<typename T> void CudaRasterizer::Rasterizer::eval_jtf_and_get_sparse_jacobian(PackedGSGNDataSpec data, T* r_vec, __half** sparse_jacobians, int** index_map, float** per_gaussian_cache) {
    data.allocate_pointer_memory();
	CHECK_CUDA(CudaRasterizer::GSGN::eval_jtf_and_get_sparse_jacobian<T>(data, r_vec, sparse_jacobians, index_map, per_gaussian_cache), data.debug)
    data.free_pointer_memory();
}
template void CudaRasterizer::Rasterizer::eval_jtf_and_get_sparse_jacobian<float>(PackedGSGNDataSpec data, float* r_vec, __half** sparse_jacobians, int** index_map, float** per_gaussian_cache);
template void CudaRasterizer::Rasterizer::eval_jtf_and_get_sparse_jacobian<double>(PackedGSGNDataSpec data, double* r_vec, __half** sparse_jacobians, int** index_map, float** per_gaussian_cache);

// apply_j
template<typename T> void CudaRasterizer::Rasterizer::apply_j(PackedGSGNDataSpec data, T* x_vec, T* x_resorted_vec, T* jx_vec, __half** sparse_jacobians, int** index_map, float** per_gaussian_cache, int** segments, int** segments_to_gaussians, int** num_gaussians_in_block, int** block_offset_in_segments, int max_gaussians_per_block, int* max_gaussians_per_block_per_image_ptr) {
    // data.allocate_pointer_memory();
	CHECK_CUDA(CudaRasterizer::GSGN::apply_j<T>(data, x_vec, x_resorted_vec, jx_vec, sparse_jacobians, index_map, per_gaussian_cache, segments, segments_to_gaussians, num_gaussians_in_block, block_offset_in_segments, max_gaussians_per_block, max_gaussians_per_block_per_image_ptr), data.debug)
    // data.free_pointer_memory();
}
template void CudaRasterizer::Rasterizer::apply_j<float>(PackedGSGNDataSpec data, float* x_vec, float* x_resorted_vec, float* jx_vec, __half** sparse_jacobians, int** index_map, float** per_gaussian_cache, int** segments, int** segments_to_gaussians, int** num_gaussians_in_block, int** block_offset_in_segments, int max_gaussians_per_block, int* max_gaussians_per_block_per_image_ptr);
template void CudaRasterizer::Rasterizer::apply_j<double>(PackedGSGNDataSpec data, double* x_vec, double* x_resorted_vec, double* jx_vec, __half** sparse_jacobians, int** index_map, float** per_gaussian_cache, int** segments, int** segments_to_gaussians, int** num_gaussians_in_block, int** block_offset_in_segments, int max_gaussians_per_block, int* max_gaussians_per_block_per_image_ptr);

// apply_jt
template<typename T> void CudaRasterizer::Rasterizer::apply_jt(PackedGSGNDataSpec data, T* g_vec, T* jx_vec, __half** sparse_jacobians, int** index_map, float** per_gaussian_cache, int** segments, int** segments_to_gaussians, int** num_gaussians_in_block, int** block_offset_in_segments, int max_gaussians_per_block, int* max_gaussians_per_block_per_image_ptr) {
    // data.allocate_pointer_memory();
	CHECK_CUDA(CudaRasterizer::GSGN::apply_jt<T>(data, g_vec, jx_vec, sparse_jacobians, index_map, per_gaussian_cache, segments, segments_to_gaussians, num_gaussians_in_block, block_offset_in_segments, max_gaussians_per_block, max_gaussians_per_block_per_image_ptr), data.debug)
    // data.free_pointer_memory();
}
template void CudaRasterizer::Rasterizer::apply_jt<float>(PackedGSGNDataSpec data, float* g_vec, float* jx_vec, __half** sparse_jacobians, int** index_map, float** per_gaussian_cache, int** segments, int** segments_to_gaussians, int** num_gaussians_in_block, int** block_offset_in_segments, int max_gaussians_per_block, int* max_gaussians_per_block_per_image_ptr);
template void CudaRasterizer::Rasterizer::apply_jt<double>(PackedGSGNDataSpec data, double* g_vec, double* jx_vec, __half** sparse_jacobians, int** index_map, float** per_gaussian_cache, int** segments, int** segments_to_gaussians, int** num_gaussians_in_block, int** block_offset_in_segments, int max_gaussians_per_block, int* max_gaussians_per_block_per_image_ptr);

// calc_preconditioner
template<typename T> void CudaRasterizer::Rasterizer::calc_preconditioner(PackedGSGNDataSpec data, T* M_vec, __half** sparse_jacobians, int** index_map, float** per_gaussian_cache, int** segments, int** segments_to_gaussians, int** num_gaussians_in_block, int** block_offset_in_segments, int max_gaussians_per_block) {
    // data.allocate_pointer_memory();
	CHECK_CUDA(CudaRasterizer::GSGN::calc_preconditioner<T>(data, M_vec, sparse_jacobians, index_map, per_gaussian_cache, segments, segments_to_gaussians, num_gaussians_in_block, block_offset_in_segments, max_gaussians_per_block), data.debug)
    // data.free_pointer_memory();
}
template void CudaRasterizer::Rasterizer::calc_preconditioner(PackedGSGNDataSpec data, float* M_vec, __half** sparse_jacobians, int** index_map, float** per_gaussian_cache, int** segments, int** segments_to_gaussians, int** num_gaussians_in_block, int** block_offset_in_segments, int max_gaussians_per_block);
template void CudaRasterizer::Rasterizer::calc_preconditioner(PackedGSGNDataSpec data, double* M_vec, __half** sparse_jacobians, int** index_map, float** per_gaussian_cache, int** segments, int** segments_to_gaussians, int** num_gaussians_in_block, int** block_offset_in_segments, int max_gaussians_per_block);

// sort_sparse_jacobians
template<typename T> void CudaRasterizer::Rasterizer::sort_sparse_jacobians(PackedGSGNDataSpec data, T** in_sparse_jacobians, T** out_sparse_jacobians, int64_t** indices) {
	// data.allocate_pointer_memory();
    CHECK_CUDA(CudaRasterizer::GSGN::sort_sparse_jacobians<T>(data, in_sparse_jacobians, out_sparse_jacobians, indices), data.debug)
    // data.free_pointer_memory();
}
template void CudaRasterizer::Rasterizer::sort_sparse_jacobians<float>(PackedGSGNDataSpec data, float** in_sparse_jacobians, float** out_sparse_jacobians, int64_t** indices);
template void CudaRasterizer::Rasterizer::sort_sparse_jacobians<double>(PackedGSGNDataSpec data, double** in_sparse_jacobians, double** out_sparse_jacobians, int64_t** indices);
template void CudaRasterizer::Rasterizer::sort_sparse_jacobians<__half>(PackedGSGNDataSpec data, __half** in_sparse_jacobians, __half** out_sparse_jacobians, int64_t** indices);
